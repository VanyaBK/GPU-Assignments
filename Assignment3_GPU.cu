#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void update(int *matrix,int *query,int *row,int *row_ele,int *no_query,int *prev_array,int n,int m,int q)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id<(m*q))
	{

	int query_no = id/m;
	int row_no = id%m;
	if(matrix[row_no*n+row[query_no]-1]==row_ele[query_no])   // Updating if the row element matches
	{	
		for(int i=0;i<no_query[query_no];i++)
		{
			if(query[prev_array[query_no]+i*3+2]==0)
			{
				atomicSub(&matrix[row_no*n+query[prev_array[query_no]+i*3]-1] , query[prev_array[query_no]+i*3+1]);
			}
			else
			{
				atomicAdd(&matrix[row_no*n+query[prev_array[query_no]+i*3]-1] , query[prev_array[query_no]+i*3+1]);
			}	
		}
	}
	}	
}

int main(int argc,char **argv)
{
  int M,N,q;
  FILE *fpi,*fpo;
  fpi=fopen(argv[1],"r");
  fpo=fopen(argv[2],"w");
  fscanf(fpi,"%d %d", &M,&N);
  int *matrix, *hmatrix;
  char character;int *row,*row_ele,*drow,*drow_ele;
  int *query,*no_query,*prev_array,*dquery,*dno_query,*dprev_array;
  hipMalloc(&matrix, (M) * (N) * sizeof(int));
  hmatrix = (int *)malloc(M * N * sizeof(int));
  for (int ii = 0; ii < M; ++ii) 
  {
    for (int jj = 0; jj < N; ++jj) 
    {
      fscanf(fpi,"%d",&hmatrix[ii*N+jj]);
    }
  }
  hipMemcpy(matrix, hmatrix, M * N * sizeof(int), hipMemcpyHostToDevice);
  fscanf(fpi,"%d", &q);
  hipMalloc(&dquery, 90 * q * sizeof(int));
  hipMalloc(&drow, q * sizeof(int));
  hipMalloc(&drow_ele, q * sizeof(int));
  hipMalloc(&dno_query, q * sizeof(int));
  hipMalloc(&dprev_array, q * sizeof(int));
  query = (int *)malloc(90 * q * sizeof(int));
  row = (int *)malloc(q * sizeof(int));
  row_ele = (int *)malloc(q * sizeof(int));
  no_query = (int *)malloc(q * sizeof(int));
  prev_array = (int *)malloc(q * sizeof(int));
  int prev=0;char c1[50];

  // Parsing Queries
  for (int i = 0; i < q; i++)
  {
  	fscanf(fpi,"%[^U]s",c1);
  	fscanf(fpi,"%c",&character);
  	fscanf(fpi," %c",&character);
  	fscanf(fpi,"%d %d %d",&row[i],&row_ele[i],&no_query[i]);
  	for(int j=0;j<no_query[i];j++)
  	{
  		fscanf(fpi," %c %d %d %c",&character,&query[prev+(j*3)],&query[prev+(j*3)+1],&character);
  		if(character=='+')
  			query[prev+(j*3)+2]=1;
  		else
  			query[prev+(j*3)+2]=0;
  	}
  	prev_array[i]=prev;
  	prev += no_query[i]*3;
  }
  hipMemcpy(dquery, query, 90 * q * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(drow, row,  q * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(drow_ele, row_ele, q * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dno_query, no_query, q * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dprev_array, prev_array, q * sizeof(int), hipMemcpyHostToDevice);
  update<<<3000,1024>>>(matrix,dquery,drow,drow_ele,dno_query,dprev_array,N,M,q);
  hipMemcpy(hmatrix, matrix, M * N * sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int ii = 0; ii < M; ++ii) 
  {
    for (int jj = 0; jj < N; ++jj) 
    {
    	if(jj==N-1)
    		fprintf(fpo,"%d ",hmatrix[ii*N+jj]);
    	else
      		fprintf(fpo,"%d ",hmatrix[ii*N+jj]);
    }
    fprintf(fpo,"\n");
  }
  return 0;
}