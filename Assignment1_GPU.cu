#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include "kernels.h"

__global__ void per_row_kernel(int *in,int N){
  int row = threadIdx.x * blockDim.y + threadIdx.y + blockIdx.x * blockDim.x * blockDim.y;
  if(row<N)
  {
    for(int i=0;i<N;i++)
    {
      if(i>row)
      {
        in[N*row + i] = in[i*N + row];
        in[i*N + row] = 0;
      }
    }
  }
}

__global__ void per_element_kernel(int *in, int N){
  long int ele = (blockIdx.x*gridDim.y+blockIdx.y)*(gridDim.z*blockDim.x)+(blockIdx.z*blockDim.x+threadIdx.x);
  if(ele < N*N-1)
  {
    int x = ele/N;
    int y = ele%N;
    if(ele > x*N+x)
    {
      in[ele] = in[y*N+x];
      in[y*N+x] = 0;
    }
  }	
}

__global__ void per_element_kernel_2D(int *in, int N){
  long int ele = (blockIdx.x*gridDim.y+blockIdx.y)*(blockDim.x*blockDim.y)+(threadIdx.x*blockDim.y+threadIdx.y);
  if(ele < N*N-1)
  {
    int x = ele/N;
    int y = ele%N;
    if(ele > x*N+x)
    {
      in[ele] = in[y*N+x];
      in[y*N+x] = 0;
    }
  }
}
