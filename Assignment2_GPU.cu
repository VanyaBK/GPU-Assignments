#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void sumRandC(int* A, int* B, int m, int n, int p, int q, int k)
{
  int id=blockIdx.x*blockDim.x + threadIdx.x,idx;
  if(id<((m*n)/k))
  {
  for(int i=0;i<k;i++)
  {
    idx = id+i*((m*n)/k);
    B[idx+(idx/n)] = A[idx];
    atomicAdd(&B[(((idx/n)+1)*n)+(idx/n)],A[idx]);  // Adds elements to the row end
    atomicAdd(&B[(m*n)+m+(idx%n)],A[idx]); // Adds elements to the column end
    if(idx==0)
      B[p*q-1] = INT_MAX;
  }
  }
}
__global__ void findMIn( int* A, int* B, int m, int n, int p, int q, int k)
{
  int id=blockIdx.x*blockDim.x + threadIdx.x,idx;
  if(id<((m*n)/k))
  {
  for(int i=0;i<k;i++)
  {
    idx=id+i*((m*n)/k);
    atomicMin(&B[p*q-1],B[(((idx/n)+1)*n)+(idx/n)]); // Checks minimum of row end elements
    atomicMin(&B[p*q-1],B[(m*n)+m+(idx%n)]); // Checks minimum of column end elements
  }
  }
}
__global__ void updateMin( int* A, int* B, int m, int n, int p, int q, int k)
{
  int id=blockIdx.x*blockDim.x + threadIdx.x,idx;
  if(id<((m*n)/k))
  {
  for(int i=0;i<k;i++)
  {
    idx = id+i*((m*n)/k)+((id+i*((m*n)/k))/n);
    if(idx%q!=n && idx/q!=m)
    { 
      atomicAdd(&B[idx],B[p*q-1]); // Adds minimum to all the elements not in the last row and column
    }
  }
  }
}
int main() 
{ 
  int M,N,k;
  scanf( "%d %d %d", &M,&N,&k);
  int *matrix,*matrix1, *hmatrix,*h1matrix;
  hipMalloc(&matrix, (M) * (N) * sizeof(int));
  hipMalloc(&matrix1, (M+1) * (N+1) * sizeof(int));
  hmatrix = (int *)malloc(M * N * sizeof(int));
  h1matrix = (int *)malloc((M+1) * (N+1) * sizeof(int));
  for (int ii = 0; ii < M; ++ii) 
  {
    for (int jj = 0; jj < N; ++jj) 
    {
      scanf("%d",&hmatrix[ii*N+jj]);
    }
  }
  hipMemcpy(matrix, hmatrix, M * N * sizeof(int), hipMemcpyHostToDevice);
  sumRandC<<<ceil((float)(M*N)/(k*1024)),1024>>>(matrix,matrix1,M,N,M+1,N+1,k);
  findMIn<<<ceil((float)(M*N)/(k*1024)),1024>>>(matrix,matrix1,M,N,M+1,N+1,k);
  updateMin<<<ceil((float)(M*N)/(k*1024)),1024>>>(matrix,matrix1,M,N,M+1,N+1,k);
  hipDeviceSynchronize();
  hipMemcpy(h1matrix, matrix1, (M+1) * (N+1) * sizeof(int), hipMemcpyDeviceToHost);
  for (int ii = 0; ii < M+1; ++ii) 
  {
    for (int jj = 0; jj < N+1; ++jj) 
    {
      printf("%d ",h1matrix[ii*(N+1)+jj]);
    } 
    printf("\n");
  }
  return 0;
}
